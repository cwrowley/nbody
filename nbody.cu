#include "hip/hip_runtime.h"
#include <omp.h>
#include <stdio.h>
#include <nvToolsExt.h>

// Macro for checking cuda errors following a cuda launch or api call
#define cudaCheckError() {                                       \
 hipError_t e=hipGetLastError();                               \
 if(e!=hipSuccess) {                                            \
   printf("Cuda failure %s:%d: '%s'\n",                          \
          __FILE__,__LINE__,hipGetErrorString(e));              \
   exit(0);                                                      \
 }                                                               \
}

__device__ __host__
inline float2 induced_velocity_single(float2 pos, float4 vort) {
  // vortex strength is vort.z
  const float eps = 1.e-6;
  float dx = pos.x - vort.x;
  float dy = pos.y - vort.y;
  float fac = vort.z / (dx * dx + dy * dy + eps);
  float2 vel = {dy * fac, -dx * fac};
  return vel;
}

void induced_vel_reference(const float2 *pos,
                           const float4 *vort,
                           float2 *vel,
                           const int N) {
  memset(vel, 0, N * sizeof(*vel));
  for (int i = 0; i < N; ++i) {   // i indexes position
    for (int j = 0; j < N; ++j) { // j indexes vortices
      float2 v = induced_velocity_single(pos[i], vort[j]);
      vel[i].x += v.x;
      vel[i].y += v.y;
    }
  }
}

void induced_vel_omp(const float2 *pos,
                     const float4 *vort,
                     float2 *vel,
                     const int N) {
  // set number of threads with
  // export OMP_NUM_THREADS=6
  memset(vel, 0, N * sizeof(*vel));
  #pragma omp parallel for
  for (int i = 0; i < N; ++i) {   // i indexes position
    #pragma omp parallel for
    for (int j = 0; j < N; ++j) { // j indexes vortices
      float2 v = induced_velocity_single(pos[i], vort[j]);
      vel[i].x += v.x;
      vel[i].y += v.y;
    }
  }
}

__global__ void induced_vel_kernel(const float2 * __restrict__ pos,
                     const float4 * __restrict__ vort,
                     float2 *vel_out,
                     const int N) {
  const int i = blockIdx.x * blockDim.x + threadIdx.x;
  const float2 p = pos[i];
  float2 vel = {0.,0.};
  for (int j = 0; j < N; ++j) {
    float2 v = induced_velocity_single(p, vort[j]);
    vel.x += v.x;
    vel.y += v.y;
  }
  vel_out[i] = vel;
}

const int TILE_SIZE = 128;

__global__
void induced_vel_kernel_smem(const float2 *pos,
                     const float4 *vort,
                     float2 *vel_out,
                     const int N) {
  __shared__ float4 smem[TILE_SIZE];
  const int i = blockIdx.x * blockDim.x + threadIdx.x;
  const int t = threadIdx.x;

  const float2 p = pos[i];
  float2 vel = {0.,0.};

  for (int j = t; j < N; j += TILE_SIZE) {
    // load a chunk of vortices into shared memory
    smem[t] = vort[j];
    __syncthreads();

    // compute contributions from each vortex in the chunk
    for (int k = 0; k < TILE_SIZE && t + blockIdx.x * TILE_SIZE < N; ++k) {
      float2 v = induced_velocity_single(p, smem[k]);
      vel.x += v.x;
      vel.y += v.y;
    }
    __syncthreads();
  }

  vel_out[i] = vel;
}

__global__
void induced_vel_kernel2(const float2 * __restrict__ pos,
                              const float4 * __restrict__ vort,
                              float2 *vel_out,
                              const int N) {
  // __shared__ float2 smem[TILE_SIZE];

  for (int i = blockIdx.y * blockDim.y + threadIdx.y; i < N; i += blockDim.y * gridDim.y) {
    // i indexes position
    // load positions into shared memory
    // smem[threadIdx.x] = pos[blockIdx.y * blockDim.y + threadIdx.x];
    // __syncthreads();

    float2 vel = {0.0, 0.0};
    float2 p = pos[i];
    // float2 p = smem[threadIdx.y];

    for (int j = blockIdx.x * blockDim.x + threadIdx.x; j < N; j += blockDim.x * gridDim.x) {
      // j indexes vortices
      float2 v = induced_velocity_single(p, vort[j]);
      vel.x += v.x;
      vel.y += v.y;
    }
    atomicAdd(&vel_out[i].x, vel.x);
    atomicAdd(&vel_out[i].y, vel.y);
    // __syncthreads();
  }
}

void induced_vel_gpu(const float2 *pos,
                     const float4 *vort,
                     float2 *vel,
                     const int N) {
  dim3 threads(TILE_SIZE);
  dim3 blocks((N + threads.x - 1)/threads.x);
  induced_vel_kernel_smem<<<blocks, threads>>>(pos, vort, vel, N);
  cudaCheckError();
}

void induced_vel_gpu2(const float2 *pos,
                      const float4 *vort,
                      float2 *vel,
                      const int N) {
  dim3 threads(32, 32);
  dim3 blocks(1, (N + threads.y - 1) / threads.y);
  // dim3 threads(N, N);
  // dim3 blocks(1, 1);
  memset(vel, 0, N * sizeof(float2));
  induced_vel_kernel2<<<blocks, threads>>>(pos, vort, vel, N);
  cudaCheckError();
}

const int THREADS_X = 8;
const int NUM_ELEMS_PER_THREAD = 32 / THREADS_X;

__global__
void induced_vel_kernel3(const float2 * __restrict__ pos,
                         const float4 * __restrict__ vort,
                         float2 *vel_out,
                         const int N) {
  const float eps = 1.e-6;

  int i = blockIdx.y * blockDim.y + threadIdx.y;
  int j = blockIdx.x * blockDim.x * NUM_ELEMS_PER_THREAD + threadIdx.x;

  float2 vel = {0.0, 0.0};
  float2 p = pos[i];

  #pragma unroll
  for (int k = 0; k < NUM_ELEMS_PER_THREAD; ++k) {
    // j indexes vortices
    float4 v = vort[j + k * THREADS_X];
    float dx = p.x - v.x;
    float dy = p.y - v.y;
    float fac = v.z / (dx * dx + dy * dy + eps);
    vel.x +=  dy * fac;
    vel.y += -dx * fac;
  }
  atomicAdd(&vel_out[i].x, vel.x);
  atomicAdd(&vel_out[i].y, vel.y);
}

void induced_vel_gpu3(const float2 *pos,
                      const float4 *vort,
                      float2 *vel,
                      const int N) {
  dim3 threads(THREADS_X, 32);
  dim3 blocks((N + 31 - 1) / 32, (N + threads.y - 1) / threads.y);
  memset(vel, 0, N * sizeof(float2));
  induced_vel_kernel3<<<blocks, threads>>>(pos, vort, vel, N);
  cudaCheckError();
}

typedef void (*func_ptr)(const float2*, const float4*, float2*, const int);

const int NUM_REPS = 1;

inline bool close(float2 a, float2 b) {
  const float rel_tol = 1.e-6;
  return fabs((a.x-b.x) * (a.x-b.x) + (a.y - b.y) * (a.y - b.y)) <
    rel_tol * fabs(a.x * a.x + a.y * a.y);
}

void time_induced_vel(const char *label,
                      func_ptr fptr,
                      const float4 *vort,
                      float2 *vel,
                      const int N,
                      bool cuda) {
  // warm up
  float2 *pos = NULL;
  hipMallocManaged(&pos, N * sizeof(*pos));
  cudaCheckError();

  for (int i = 0; i < N; ++i) {
    pos[i].x = vort[i].x;
    pos[i].y = vort[i].y;
  }
  fptr(pos, vort, vel, N);

  if (cuda) hipDeviceSynchronize();
  double start = omp_get_wtime();
  nvtxRangePushA(label);
  for (int i = 0; i < NUM_REPS; ++i) {
    fptr(pos, vort, vel, N);
  }
  if (cuda) hipDeviceSynchronize();
  nvtxRangePop();
  double end = omp_get_wtime();

  // Check the answer: return time only if answer is correct.
  float2 *validation = (float2 *) malloc(N * sizeof(float2));
  induced_vel_reference(pos, vort, validation, N);
  for (int i = 0; i < N; ++i) {
    if (!close(vel[i], validation[i])) {
      printf("%s: Error: velocity is incorrect at index %d\n", label, i);
      printf("  expected (%f, %f)\n       got (%f, %f)\n",
             validation[i].x, validation[i].y, vel[i].x, vel[i].y);
      return;
    }
  }
  free(validation);
  hipFree(pos);

  double time = (end - start) / ((double) NUM_REPS);
  double Mflops = (double) (6. * N * N) / (1000 * 1000 * 1000 * time);
  printf("%s: %f GFlops\n", label, Mflops);
}


int main() {
  // const int N = 8192;
  const int N = 1<<13;
  // const int N = 1024;
  float4 *vort = NULL;
  float2 *vel = NULL;

  hipSetDevice(1);
  cudaCheckError();

  hipMallocManaged(&vort, N * sizeof(*vort));
  hipMallocManaged(&vel, N * sizeof(*vel));
  cudaCheckError();

  // initialize vortex positions and strengths to random values
  for (int i = 0; i < N; ++i) {
    vort[i].x = rand() / (float) RAND_MAX;
    vort[i].y = rand() / (float) RAND_MAX;
    vort[i].z = rand() / (float) RAND_MAX;
    vort[i].w = 0.;
  }
  memset(vel, 0, N * sizeof(float2));

  printf("N = %d\n", N);
  time_induced_vel((char *)"CPU", induced_vel_reference, vort, vel, N, false);
  time_induced_vel((char *)"CPU + OMP", induced_vel_omp, vort, vel, N, false);
  time_induced_vel((char *)"GPU", induced_vel_gpu, vort, vel, N, true);
  time_induced_vel((char *)"GPU v2", induced_vel_gpu2, vort, vel, N, true);
  time_induced_vel((char *)"GPU v3", induced_vel_gpu3, vort, vel, N, true);

  hipFree(vort);
  hipFree(vel);
  hipDeviceReset();
  return 0;
}
