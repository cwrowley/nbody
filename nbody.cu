#include "hip/hip_runtime.h"
#include <omp.h>
#include <stdio.h>
#include <nvToolsExt.h>

// Macro for checking cuda errors following a cuda launch or api call
#define cudaCheckError() {                                       \
 hipError_t e=hipGetLastError();                               \
 if(e!=hipSuccess) {                                            \
   printf("Cuda failure %s:%d: '%s'\n",                          \
          __FILE__,__LINE__,hipGetErrorString(e));              \
   exit(0);                                                      \
 }                                                               \
}

inline float2 induced_velocity_single(float2 pos, float4 vort) {
  // vortex strength is vort.z
  const float eps = 1.e-6;
  const float2 r = {pos.x - vort.x, pos.y - vort.y};
  float rsq = r.x * r.x + r.y * r.y + eps;
  float2 vel = {vort.z * r.y / rsq, -vort.z * r.x / rsq};
  return vel;
}

void induced_vel_reference(const float2 *pos,
                           const float4 *vort,
                           float2 *vel,
                           const int N) {
  memset(vel, 0, N * sizeof(*vel));
  for (int i = 0; i < N; ++i) {   // i indexes position
    for (int j = 0; j < N; ++j) { // j indexes vortices
      float2 v = induced_velocity_single(pos[i], vort[j]);
      vel[i].x += v.x;
      vel[i].y += v.y;
    }
  }
}

void induced_vel_omp(const float2 *pos,
                     const float4 *vort,
                     float2 *vel,
                     const int N) {
  // set number of threads with
  // export OMP_NUM_THREADS=6
  memset(vel, 0, N * sizeof(*vel));
  #pragma omp parallel for
  for (int i = 0; i < N; ++i) {   // i indexes position
    #pragma omp parallel for
    for (int j = 0; j < N; ++j) { // j indexes vortices
      float2 v = induced_velocity_single(pos[i], vort[j]);
      vel[i].x += v.x;
      vel[i].y += v.y;
    }
  }
}

__global__ void induced_vel_kernel(const float2 *pos,
                     const float4 *vort,
                     float2 *vel_out,
                     const int N) {
  const int i = blockIdx.x * blockDim.x + threadIdx.x;
  const float2 p = pos[i];
  float2 vel = {0.,0.};
  for (int j = 0; j < N; ++j) {
    const float eps = 1.e-6;
    const float2 r = {p.x - vort[j].x, p.y - vort[j].y};
    float fac = vort[j].z / (r.x * r.x + r.y * r.y + eps);
    vel.x +=  r.y * fac;
    vel.y += -r.x * fac;
  }
  vel_out[i] = vel;
}

void induced_vel_gpu(const float2 *pos,
                     const float4 *vort,
                     float2 *vel,
                     const int N) {
  dim3 threads(128);
  dim3 blocks((N + threads.x - 1)/threads.x);
  induced_vel_kernel<<<blocks, threads>>>(pos, vort, vel, N);
  cudaCheckError();
}


typedef void (*func_ptr)(const float2*, const float4*, float2*, const int);

const int NUM_REPS = 1;

inline bool close(float2 a, float2 b) {
  const float rel_tol = 1.e-6;
  return fabs((a.x-b.x) * (a.x-b.x) + (a.y - b.y) * (a.y - b.y)) <
    rel_tol * fabs(a.x * a.x + a.y * a.y);
}

void time_induced_vel(const char *label,
                      func_ptr fptr,
                      const float4 *vort,
                      float2 *vel,
                      const int N,
                      bool cuda) {
  // warm up
  float2 *pos = NULL;
  hipMallocManaged(&pos, N * sizeof(*pos));
  cudaCheckError();

  for (int i = 0; i < N; ++i) {
    pos[i].x = vort[i].x;
    pos[i].y = vort[i].y;
  }
  fptr(pos, vort, vel, N);

  if (cuda) hipDeviceSynchronize();
  double start = omp_get_wtime();
  nvtxRangePushA(label);
  for (int i = 0; i < NUM_REPS; ++i) {
    fptr(pos, vort, vel, N);
  }
  if (cuda) hipDeviceSynchronize();
  nvtxRangePop();
  double end = omp_get_wtime();

  // Check the answer: return time only if answer is correct.
  float2 *validation = (float2 *) malloc(N * sizeof(float2));
  induced_vel_reference(pos, vort, validation, N);
  for (int i = 0; i < N; ++i) {
    if (!close(vel[i], validation[i])) {
      printf("%s: Error: velocity is incorrect at index %d\n", label, i);
      printf("  expected (%f, %f)\n       got (%f, %f)\n",
             validation[i].x, validation[i].y, vel[i].x, vel[i].y);
      return;
    }
  }
  free(validation);
  hipFree(pos);

  double time = (end - start) / ((double) NUM_REPS);
  double Mflops = (double) (6 * N * N) / (1000 * 1000 * 1000 * time);
  printf("%s: %f GFlops\n", label, Mflops);
}


int main() {
  const int N = 8192;
  float4 *vort = NULL;
  float2 *vel = NULL;

  hipMallocManaged(&vort, N * sizeof(*vort));
  hipMallocManaged(&vel, N * sizeof(*vel));
  cudaCheckError();

  // initialize vortex positions and strengths to random values
  for (int i = 0; i < N; ++i) {
    vort[i].x = rand() / (float) RAND_MAX;
    vort[i].y = rand() / (float) RAND_MAX;
    vort[i].z = rand() / (float) RAND_MAX;
    vort[i].w = 0.;
  }
  memset(vel, 0, N * sizeof(float2));

  time_induced_vel((char *)"CPU", induced_vel_reference, vort, vel, N, false);
  time_induced_vel((char *)"CPU + OMP", induced_vel_omp, vort, vel, N, false);
  time_induced_vel((char *)"GPU", induced_vel_gpu, vort, vel, N, true);

  hipFree(vort);
  hipFree(vel);
  return 0;
}
