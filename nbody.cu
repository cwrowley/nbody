#include "hip/hip_runtime.h"
#include <omp.h>
#include <stdio.h>
#include <nvToolsExt.h>

// Macro for checking cuda errors following a cuda launch or api call
#define cudaCheckError() {                                       \
 hipError_t e=hipGetLastError();                               \
 if(e!=hipSuccess) {                                            \
   printf("Cuda failure %s:%d: '%s'\n",                          \
          __FILE__,__LINE__,hipGetErrorString(e));              \
   exit(0);                                                      \
 }                                                               \
}

inline float2 induced_velocity_single(float2 pos, float2 vort, float gam) {
  const float eps = 1.e-6;
  const float2 r = {pos.x - vort.x, pos.y - vort.y};
  float rsq = r.x * r.x + r.y * r.y + eps;
  float2 vel = {gam * r.x / rsq, -gam * r.y / rsq};
  return vel;
}

void induced_vel_reference(const float2 *pos,
                           const float2 *vort,
                           const float *gam,
                           float2 *vel,
                           const int N) {
  memset(vel, 0, N * sizeof(float2));
  for (int i = 0; i < N; ++i) {   // i indexes position
    for (int j = 0; j < N; ++j) { // j indexes vortices
      float2 v = induced_velocity_single(pos[i], vort[j], gam[j]);
      vel[i].x += v.x;
      vel[i].y += v.y;
    }
  }
}

void induced_vel_omp(const float2 *pos,
                     const float2 *vort,
                     const float *gam,
                     float2 *vel,
                     const int N) {
  // set number of threads with
  // export OMP_NUM_THREADS=6
  memset(vel, 0, N * sizeof(float2));
  #pragma omp parallel for
  for (int i = 0; i < N; ++i) {   // i indexes position
    #pragma omp parallel for
    for (int j = 0; j < N; ++j) { // j indexes vortices
      float2 v = induced_velocity_single(pos[i], vort[j], gam[j]);
      vel[i].x += v.x;
      vel[i].y += v.y;
    }
  }
}

__global__ void induced_vel_kernel(const float2 *pos,
                     const float2 *vort,
                     const float *gam,
                     float2 *vel_out,
                     const int N) {
  const int i = blockIdx.x * blockDim.x + threadIdx.x;
  const float2 p = pos[i];
  float2 vel = {0.,0.};
  for (int j = 0; j < N; ++j) {
    const float eps = 1.e-6;
    const float2 r = {p.x - vort[j].x, p.y - vort[j].y};
    float rsq = r.x * r.x + r.y * r.y + eps;
    vel.x +=  gam[j] * r.x / rsq;
    vel.y += -gam[j] * r.y / rsq;
  }
  vel_out[i] = vel;
}

void induced_vel_gpu(const float2 *pos,
                     const float2 *vort,
                     const float *gam,
                     float2 *vel,
                     const int N) {
  dim3 threads(128);
  dim3 blocks((N + threads.x - 1)/threads.x);
  induced_vel_kernel<<<blocks, threads>>>(pos, vort, gam, vel, N);
  cudaCheckError();
}


typedef void (*func_ptr)(const float2*, const float2*, const float*, float2*, const int);

const int NUM_REPS = 1;

void time_induced_vel(const char *label,
                      func_ptr fptr,
                      const float2 *vort,
                      const float *gam,
                      float2 *vel,
                      const int N,
                      bool cuda) {
  // warm up
  fptr(vort, vort, gam, vel, N);

  if (cuda) hipDeviceSynchronize();
  double start = omp_get_wtime();
  nvtxRangePushA(label);
  for (int i = 0; i < NUM_REPS; ++i) {
    fptr(vort, vort, gam, vel, N);
  }
  if (cuda) hipDeviceSynchronize();
  nvtxRangePop();
  double end = omp_get_wtime();

  // Check the answer: return time only if answer is correct.
  float2 *validation = (float2 *) malloc(N * sizeof(float2));
  induced_vel_reference(vort, vort, gam, validation, N);
  for (int i = 0; i < N; ++i) {
    if (vel[i].x != validation[i].x || vel[i].y != validation[i].y) {
      printf("%s: Error: velocity is incorrect at index %d\n", label, i);
      return;
    }
  }
  free(validation);

  double time = (end - start) / ((double) NUM_REPS);
  double Mflops = (double) (6 * N * N) / (1000 * 1000 * 1000 * time);
  printf("%s: %f GFlops\n", label, Mflops);
}


int main() {
  const int N = 8192;
  float2 *vort = NULL;
  float2 *vel = NULL;
  float *gam = NULL;

  hipMallocManaged(&vort, N * sizeof(float2));
  hipMallocManaged(&vel, N * sizeof(float2));
  hipMallocManaged(&gam, N * sizeof(float));
  cudaCheckError();

  // initialize vortex positions and strengths to random values
  for (int i = 0; i < N; ++i) {
    vort[i].x = rand() / (float) RAND_MAX;
    vort[i].y = rand() / (float) RAND_MAX;
    gam[i] = rand() / (float) RAND_MAX;
  }
  memset(vel, 0, N * sizeof(float2));

  time_induced_vel((char *)"CPU", induced_vel_reference, vort, gam, vel, N, false);
  time_induced_vel((char *)"CPU + OMP", induced_vel_omp, vort, gam, vel, N, false);
  time_induced_vel((char *)"GPU", induced_vel_gpu, vort, gam, vel, N, true);

  hipFree(vort);
  hipFree(vel);
  hipFree(gam);
  return 0;
}
